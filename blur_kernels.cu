#include "hip/hip_runtime.h"
#include "./gaussian_kernel.h" 


#define BLOCK 32
#define TILE_WIDTH BLOCK

/*
The actual gaussian blur kernel to be implemented by 
you. Keep in mind that the kernel operates on a 
single channel.
 */

//Shared memory kernel
__global__ 
void gaussianBlur_shared(unsigned char *d_in, unsigned char *d_out, 
        const int rows, const int cols, float *d_filter, const int filterWidth){
  
  //shared memory array
  __shared__ unsigned char local_pixels[TILE_WIDTH + (filterWidth/2)*2][TILE_WIDTH + (filterWidth/2)*2];
  __shared__ float local_filter[filterWidth][filterWidth];
  
  // Global Image id
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  
  // Shared Data id
  int local_x = threadIdx.x + (filterWidth/2);
  int local_y = threadIdx.y + (filterWidth/2);

  
  // loading image into shared memory using the 1st thread.
  if(threadIdx.x == 0 && threadIdx.y == 0) {
    for (int i=0; i < TILE_WIDTH + (filterWidth/2)*2; i++){
        for (int j=0; j < TILE_WIDTH + (filterWidth/2)*2; j++){
            if ((py + i - (filterWidth/2)) < rows && (py + i - (filterWidth/2)) > -1 && (px + j - (filterWidth/2)) < cols && (px + j - (filterWidth/2)) > -1){
                local_pixels[i][j] = d_in[(py + i - (filterWidth/2)) * numCols + (px + j - (filterWidth/2))];
            }
        }
    }
    // Loading the filter in the shared memory using the 1st thread.
    for (int i=0; i < filterWidth; i++){
        for (int j=0; j < filterWidth; j++){
            local_filter[i][j] = d_filter[i][j];
        }
    }
  }

  __syncthreads();

  if (px < cols && py < rows) {
    float pixval = 0.0;
    for(int blurRow = -(filterWidth / 2); blurRow < (filterWidth / 2) + 1; ++blurRow) {
        for(int blurCol = -(filterWidth / 2); blurCol < (filterWidth / 2) + 1; ++blurCol) {        
            int curRow = py + blurRow;
            int curCol = px + blurCol;
            if(curRow > -1 && curRow < rows && curCol > -1 && curCol < cols) {
                pixval += ((float) local_pixels[local_y + blurRow][local_x + blurCol] * local_filter[(blurRow + (filterWidth/2))][(blurCol + filterWidth/2)]);
                //pixval += ((float) d_in[curRow * cols + curCol] * d_filter[(blurRow + (filterWidth/2)) * filterWidth + (blurCol + filterWidth/2)]);
            }
        }
    }

    __syncthreads();
    d_out[py * cols + px] = (unsigned char) pixval;
  }
} 
















__global__ 
void gaussianBlur(unsigned char *d_in, unsigned char *d_out, 
        const int rows, const int cols, float *d_filter, const int filterWidth){
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  int i = py * cols + px;
  if (px < cols && py < rows) {
    float pixval = 0.0;
    for(int blurRow = -(filterWidth / 2); blurRow < (filterWidth / 2) + 1; ++blurRow) {
        for(int blurCol = -(filterWidth / 2); blurCol < (filterWidth / 2) + 1; ++blurCol) {        
            int curRow = py + blurRow;
            int curCol = px + blurCol;
            if(curRow > -1 && curRow < rows && curCol > -1 && curCol < cols) {
                pixval += ((float) d_in[curRow * cols + curCol] * d_filter[(blurRow + (filterWidth/2))*filterWidth + (blurCol + filterWidth/2)]);
            }
        }
    }
    d_out[i] = (unsigned char) pixval;
  }
} 



/*
  Given an input RGBA image separate 
  that into appropriate rgba channels.
 */
__global__ 
void separateChannels(uchar4 *d_imrgba, unsigned char *d_r, unsigned char *d_g, unsigned char *d_b,
        const int rows, const int cols){

  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  if (px < cols && py < rows) {
    int i = py * cols + px;
    d_r[i] = d_imrgba[i].x;
    d_g[i] = d_imrgba[i].y;
    d_b[i] = d_imrgba[i].z;
  }
} 
 

/*
  Given input channels combine them 
  into a single uchar4 channel. 

  You can use some handy constructors provided by the 
  cuda library i.e. 
  make_int2(x, y) -> creates a vector of type int2 having x,y components 
  make_uchar4(x,y,z,255) -> creates a vector of uchar4 type x,y,z components 
  the last argument being the transperency value. 
 */
__global__ 
void recombineChannels(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, uchar4 *d_orgba,
        const int rows, const int cols){
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  if (px < cols && py < rows) {
    int i = py * cols + px;
    d_orgba[i] = make_uchar4(d_b[i], d_g[i], d_r[i], 255);
    //d_orgba[i].x = d_r[i];
    //d_orgba[i].y = d_g[i];
    //d_orgba[i].z = d_b[i];
  }
}

void your_gauss_blur(uchar4* d_imrgba, uchar4 *d_oimrgba, size_t rows, size_t cols, 
        unsigned char *d_red, unsigned char *d_green, unsigned char *d_blue, 
        unsigned char *d_rblurred, unsigned char *d_gblurred, unsigned char *d_bblurred,
        float *d_filter,  int filterWidth){
 


        dim3 blockSize(BLOCK, BLOCK, 1);
        dim3 gridSize((cols-1)/BLOCK + 1, (rows-1)/BLOCK + 1, 1);

        separateChannels<<<gridSize, blockSize>>>(d_imrgba, d_red, d_green, d_blue, rows, cols);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_red, d_rblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_green, d_gblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_blue, d_bblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        recombineChannels<<<gridSize, blockSize>>>(d_rblurred, d_gblurred, d_bblurred, d_oimrgba, rows, cols);

        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());   

}




